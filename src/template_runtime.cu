#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2013 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/* Template project which demonstrates the basics on how to setup a project
* example application, doesn't use cutil library.
*/

#include <stdio.h>
#include <string.h>
#include <limits.h>
#include <igraph.h>
#include <iostream>

int numNodos = 0;
#define INT_MAX2 100

using namespace std;

#ifdef _WIN32
#define STRCASECMP  _stricmp
#define STRNCASECMP _strnicmp
#else
#define STRCASECMP  strcasecmp
#define STRNCASECMP strncasecmp
#endif

#define ASSERT(x, msg, retcode) \
    if (!(x)) \
    { \
        cout << msg << " " << __FILE__ << ":" << __LINE__ << endl; \
        return retcode; \
    }

__global__ void sequence_gpu(int *d_ptr, int length)
{
    int elemID = blockIdx.x * blockDim.x + threadIdx.x;

    if (elemID < length)
    {
        d_ptr[elemID] = elemID;
    }
}

__global__ void init_stp(int *d_stp, int numNodos)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;


	if (i < numNodos) {
		if (j < numNodos) {
			d_stp[i+numNodos*j] = INT_MAX2;
		}
	}

}

__global__ void init_boolean_vector(bool *d_boolean_vector, int numNodos)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i<numNodos){
		d_boolean_vector[i] = false;
	}

}

void sequence_cpu(int *h_ptr, int length)
{
    for (int elemID=0; elemID<length; elemID++)
    {
        h_ptr[elemID] = elemID;
    }
}

int print_boolean_vector(bool *vector){

	int i;
	for (i = 0; i < numNodos; ++i) {
			cout<<" "<<vector[i];
	}

	return EXIT_SUCCESS;
}

int printMatrix(int *matrix){

	int i,j;
	for (i = 0; i < numNodos; ++i) {
		for (j = 0; j < numNodos; ++j) {
			cout<<" "<<matrix[i+numNodos*j];
		}
		cout<<endl;
	}

	return EXIT_SUCCESS;
}

int stpPrim(int *grafo){
	int *h_stp;
	int *d_grafo, *d_stp;

	bool *h_boolean_vector, *d_boolean_vector;

	ASSERT(hipSuccess == hipHostMalloc(&h_boolean_vector, numNodos * sizeof(bool)), "Host allocation of "   << numNodos << " booleans failed", -1);

	ASSERT(hipSuccess == hipHostMalloc(&h_stp, numNodos*numNodos * sizeof(int)), "Host allocation of "   << numNodos*numNodos << " ints failed", -1);

	ASSERT(hipSuccess == hipMalloc(&d_grafo, numNodos*numNodos * sizeof(int)), "Device allocation of " << numNodos*numNodos << " ints failed", -1);

	ASSERT(hipSuccess == hipMalloc(&d_boolean_vector, numNodos * sizeof(bool)), "Device allocation of " << numNodos << " booleans failed", -1);

	ASSERT(hipSuccess == hipMalloc(&d_stp, numNodos*numNodos * sizeof(int)), "Device allocation of " << numNodos*numNodos << " ints failed", -1);

	ASSERT(hipSuccess == hipMemcpy(d_grafo, grafo, numNodos*numNodos*sizeof(int), hipMemcpyHostToDevice), "Copy of " << numNodos*numNodos << " ints from host to device failed", -1);

	////////inicialización STP///////////////////////////////////////////////////////////////////
	dim3 cudaBlockSize(32,32,1);
	dim3 cudaGridSize((numNodos + cudaBlockSize.x - 1) / cudaBlockSize.x, (numNodos + cudaBlockSize.y - 1) / cudaBlockSize.y, 1);
	init_stp<<<cudaGridSize, cudaBlockSize>>>(d_stp, numNodos);
	ASSERT(hipSuccess == hipGetLastError(), "Kernel launch failed", -1);
	ASSERT(hipSuccess == hipDeviceSynchronize(), "Kernel synchronization failed", -1);
	ASSERT(hipSuccess == hipMemcpy(h_stp, d_stp, numNodos*numNodos *sizeof(int), hipMemcpyDeviceToHost), "Copy of " << numNodos*numNodos << " ints from device to host failed", -1);
	//printMatrix(h_stp);
	//////////////////////////////////////////////////////////////////////////////////////////////////////


	////////inicialización BOOLEAN VECTOR///////////////////////////////////////////////////////////////////
	dim3 cudaBlockSize2(32,1,1);
	dim3 cudaGridSize2((numNodos + cudaBlockSize.x - 1) / cudaBlockSize.x, 1, 1);
	init_boolean_vector<<<cudaGridSize2, cudaBlockSize2>>>(d_boolean_vector, numNodos);
	ASSERT(hipSuccess == hipGetLastError(), "Kernel launch failed", -1);
	ASSERT(hipSuccess == hipDeviceSynchronize(), "Kernel synchronization failed", -1);
	ASSERT(hipSuccess == hipMemcpy(h_boolean_vector, d_boolean_vector, numNodos *sizeof(bool), hipMemcpyDeviceToHost), "Copy of " << numNodos*numNodos << " ints from device to host failed", -1);
	print_boolean_vector(h_boolean_vector);
	//////////////////////////////////////////////////////////////////////////////////////////////////////

	ASSERT(hipSuccess == hipHostFree(h_stp),   "Host deallocation failed",   -1);
	ASSERT(hipSuccess == hipFree(d_stp),   "Device stp deallocation failed",   -1);
	ASSERT(hipSuccess == hipFree(d_grafo),   "Device grafo deallocation failed",   -1);

	return EXIT_SUCCESS;
}

int* readGrafo(){

	int *grafo;
	igraph_matrix_t gMatrix;
	igraph_t g;
	igraph_i_set_attribute_table(&igraph_cattribute_table);
	FILE *ifile;
	ifile=fopen("/home/john/Documents/celegansneural.gml"/*"/home/john/git/primAlgorithm/grafo.gml"*/, "r");
	if (ifile==0) {
		printf("Problema abriendo archivo de grafo\n");
		return NULL;
	}
	igraph_read_graph_gml(&g, ifile);

	fclose(ifile);
	numNodos = igraph_vcount(&g);
	grafo = (int *)malloc(numNodos*numNodos*sizeof(int));
	igraph_matrix_init(&gMatrix,numNodos,numNodos);
	igraph_get_adjacency(&g,&gMatrix,IGRAPH_GET_ADJACENCY_BOTH,1);

	igraph_vector_t el;
	int ii, jj, n;
	igraph_vector_init(&el, 0);
	igraph_get_edgelist(&g, &el, 0);
	n = igraph_ecount(&g);

	memset(grafo,INT_MAX2,numNodos*numNodos*sizeof(int));

	  for (ii=0, jj=0; ii<n; ii++, jj+=2) {
	    grafo[((long)VECTOR(el)[jj])+numNodos*((long)VECTOR(el)[jj+1])] = (int)EAN(&g, "weight", ii);
	    grafo[((long)VECTOR(el)[jj+1])+numNodos*((long)VECTOR(el)[jj])] =  (int)EAN(&g, "weight", ii);
	  }

	printf("\nNumero de nodos %d\n",numNodos);

	igraph_vector_destroy(&el);
	igraph_destroy(&g);
	return grafo;

}


int main(int argc, char **argv)
{
    printf("%s Starting...\n\n", argv[0]);

    cout << "CUDA Runtime API template" << endl;
    cout << "=========================" << endl;
    cout << "Self-test started" << endl;

    const int N = 100;

    int *d_ptr;
    ASSERT(hipSuccess == hipMalloc(&d_ptr, N * sizeof(int)), "Device allocation of " << N << " ints failed", -1);

    int *h_ptr;
    ASSERT(hipSuccess == hipHostMalloc(&h_ptr, N * sizeof(int)), "Host allocation of "   << N << " ints failed", -1);

    cout << "Memory allocated successfully" << endl;

    dim3 cudaBlockSize(32,1,1);
    dim3 cudaGridSize((N + cudaBlockSize.x - 1) / cudaBlockSize.x, 1, 1);
    sequence_gpu<<<cudaGridSize, cudaBlockSize>>>(d_ptr, N);
    ASSERT(hipSuccess == hipGetLastError(), "Kernel launch failed", -1);
    ASSERT(hipSuccess == hipDeviceSynchronize(), "Kernel synchronization failed", -1);

    sequence_cpu(h_ptr, N);

    cout << "CUDA and CPU algorithm implementations finished" << endl;

    int *h_d_ptr;
    ASSERT(hipSuccess == hipHostMalloc(&h_d_ptr, N *sizeof(int)), "Host allocation of " << N << " ints failed", -1);
    ASSERT(hipSuccess == hipMemcpy(h_d_ptr, d_ptr, N *sizeof(int), hipMemcpyDeviceToHost), "Copy of " << N << " ints from device to host failed", -1);
    bool bValid = true;

    for (int i=0; i<N && bValid; i++)
    {
        if (h_ptr[i] != h_d_ptr[i])
        {
            bValid = false;
        }
    }

    ASSERT(hipSuccess == hipFree(d_ptr),       "Device deallocation failed", -1);
    ASSERT(hipSuccess == hipHostFree(h_ptr),   "Host deallocation failed",   -1);
    ASSERT(hipSuccess == hipHostFree(h_d_ptr), "Host deallocation failed",   -1);

    cout << "Memory deallocated successfully" << endl;
    cout << "TEST Results " << endl;

    int *grafo;
    grafo = readGrafo();
    stpPrim(grafo);
    free(grafo);
    exit(bValid ? EXIT_SUCCESS : EXIT_FAILURE);
}
